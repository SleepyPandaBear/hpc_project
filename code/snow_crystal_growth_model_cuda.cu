#include "hip/hip_runtime.h"
#include "math.h"
#include "stdlib.h"
#include "stdio.h"
#include "string.h"
#include "snow_crystal_growth_model.h"

// NOTE(miha): Imprt CUDA related libraries.
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime_api.h"

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image.h"
#include "stb_image_write.h"

//#define DRAW_CELL_BORDER
#include "snow_crystal_growth_renderer.h"

//#define SAVE_DURING_ITERATIONS
#define SAVE_DURING_ITERATIONS_INTERVAL (50)

b32
GenerateGrid(grid *Grid, f32 Beta)
{
    // NOTE(miha): Grid has to be odd sized.
    if(Grid->Size % 2 == 0)
    {
        printf("Grid has to be odd sized, %d mod 2 != 1\n", Grid->Size);
        return 0;
    }

    cell *Cells = (cell *)malloc(Grid->Size*Grid->Size*sizeof(cell));
    memset(Cells, 0, Grid->Size*Grid->Size*sizeof(cell));

    // NOTE(miha): Set top row cells to EDGE
    for(u32 RowIndex = 0; RowIndex < Grid->Size; ++RowIndex)
    {
        Cells[RowIndex].Type = EDGE;
    }

    // NOTE(miha): Set bottom row cells to EDGE
    for(u32 RowIndex = 0; RowIndex < Grid->Size; ++RowIndex)
    {
        //printf("bottom: row_idx: %d\n", RowIndex);
        Cells[(Grid->Size-1)*Grid->Size + RowIndex].Type = EDGE;
    }

    // NOTE(miha): Set left column cells to EDGE
    for(u32 ColumnIndex = 0; ColumnIndex < Grid->Size; ++ColumnIndex)
    {
        Cells[ColumnIndex*Grid->Size].Type = EDGE;
    }

    // NOTE(miha): Set right column cells to EDGE
    for(u32 ColumnIndex = 0; ColumnIndex < Grid->Size; ++ColumnIndex)
    {
        //printf("right: col_idx: %d\n", ColumnIndex);
        Cells[(Grid->Size-1) + ColumnIndex*Grid->Size].Type = EDGE;
    }

    // NOTE(miha): Set all cell values to Beta.
    for(u32 CellIndex = 0; CellIndex < Grid->Size*Grid->Size; ++CellIndex)
    {
        Cells[CellIndex].Value = Beta;
    }

    // NOTE(miha): Set middle cell to FROZEN
    Cells[(Grid->Size/2)*Grid->Size + (Grid->Size/2)].Type = FROZEN;
    Cells[(Grid->Size/2)*Grid->Size + (Grid->Size/2)].Value = 1.0f;

    // TODO(miha): Think about this, is it better to set BOUANDY around seed at
    // the start or should it be calculated?
#if 0
    // NOTE(miha): Set neihbours of the middle cell to BOUNDARY
    for(u32 Direction = 0; Direction < 6; ++Direction)
    {
        ivec2 Neighbour = GridNeighbour(Grid->Size/2, Grid->Size/2, Direction);
        Cells[Neighbour.Row*Grid->Size + Neighbour.Column].Type = BOUNDARY;
    }
#endif

    Grid->Cells = Cells;

    return 1;
}

#if 0
b32
IsReceptive(cell Cell)
{
    if(Cell.Type == FROZEN || Cell.Type == BOUNDARY)
        return 1;
    return 0;
}
#endif

__forceinline__ __device__ b32
gpuIsReceptive(cell Cell)
{
    if(Cell.Type == FROZEN || Cell.Type == BOUNDARY)
        return 1;
    return 0;
}

__device__ ivec2
gpuGridNeighbour(u32 Row, u32 Column, u32 Direction)
{
    i32 DirectionDifference[2][6][2] = {
        // Even column
        {{1, 1}, {1, 0}, {0, -1}, {-1, 0}, {-1, 1}, {0, 1}},
        // Odd columns
        {{1, 0}, {1, -1}, {0, -1}, {-1, -1}, {-1, 0}, {0, 1}}};
    u32 Parity = Column & 1;
    i32 *Difference = DirectionDifference[Parity][Direction];
    return ivec2{(i32)Row+Difference[1], (i32)Column+Difference[0]};
}

__global__ void
IterationGPUShared(grid CurrentGrid, grid NextGrid, u32 *MaxColumn, f32 *Alpha, f32 *Beta, f32 *Gamma)
{
    u32 LocalID = threadIdx.x;
    u32 GlobalID = blockIdx.x * blockDim.x + threadIdx.x;
    u32 Row = GlobalID / CurrentGrid.Size;
    u32 Column = GlobalID % CurrentGrid.Size;

    // NOTE(miha): BlockSize = 256
    // If Cell.Type != EDGE we can init values?
    __shared__ cell LocalCells[256 * 3 + 4];

    LocalCells[LocalID] = CurrentGrid.Cells[GlobalID];
    LocalCells[LocalID] = CurrentGrid.Cells[GlobalID];

    if(GlobalID < CurrentGrid.Size*CurrentGrid.Size)
    {
        cell Cell = CurrentGrid.Cells[GlobalID];

        if(Cell.Type == EDGE)
        {
            CurrentGrid.Cells[GlobalID].Value = *Beta;
        }
        else
        {
            f32 NewWaterValue = 0.0f;
            if(gpuIsReceptive(Cell))
            {
                NewWaterValue += Cell.Value;

                f32 NeighbourDiffusion = 0.0f;
                for(u32 Direction = 0; Direction < 6; ++Direction)
                {
                    ivec2 Neighbour = gpuGridNeighbour(Row, Column, Direction);
                    if(Neighbour.Row < 0 || Neighbour.Row > CurrentGrid.Size || Neighbour.Column < 0 || Neighbour.Column > CurrentGrid.Size)
                    {
                        // NOTE(miha): Out of bounds, ignore it.
                    }
                    else
                    {
                        cell NeighbourCell = CurrentGrid.Cells[Neighbour.Row * CurrentGrid.Size + Neighbour.Column];
                        if(!gpuIsReceptive(NeighbourCell))
                        {
                            NeighbourDiffusion += NeighbourCell.Value;
                        }
                    }
                }
                NeighbourDiffusion /= 6.0f;
                NewWaterValue += (*Alpha/2.0f)*NeighbourDiffusion;
                NewWaterValue += *Gamma;
            }
            else
            {
                NewWaterValue += Cell.Value;
                f32 NeighbourDiffusion = 0.0f;
                for(u32 Direction = 0; Direction < 6; ++Direction)
                {
                    ivec2 Neighbour = gpuGridNeighbour(Row, Column, Direction);
                    if(Neighbour.Row < 0 || Neighbour.Row > CurrentGrid.Size || Neighbour.Column < 0 || Neighbour.Column > CurrentGrid.Size)
                    {
                        // NOTE(miha): Out of bounds, ignore it.
                    }
                    else
                    {
                        cell NeighbourCell = CurrentGrid.Cells[Neighbour.Row * CurrentGrid.Size + Neighbour.Column];
                        if(!gpuIsReceptive(NeighbourCell))
                        {
                            NeighbourDiffusion += NeighbourCell.Value;
                        }
                    }
                }
                NeighbourDiffusion /= 6.0f;
                NewWaterValue += (*Alpha/2.0f)*(NeighbourDiffusion - Cell.Value);
            }

            NextGrid.Cells[GlobalID].Value = NewWaterValue;
            if(NewWaterValue > 1.0f)
            {
                NextGrid.Cells[GlobalID].Type = FROZEN;
                for(u32 Direction = 0; Direction < 6; ++Direction)
                {
                    ivec2 Neighbour = gpuGridNeighbour(Row, Column, Direction);
                    if(Neighbour.Row < 0 || Neighbour.Row > CurrentGrid.Size || Neighbour.Column < 0 || Neighbour.Column > CurrentGrid.Size)
                    {
                        // NOTE(miha): Out of bounds, ignore it.
                    }
                    else
                    {
                        cell NeighbourCell = NextGrid.Cells[Neighbour.Row * NextGrid.Size + Neighbour.Column];
                        if(NeighbourCell.Type == EDGE)
                        {
                            if(Column > *MaxColumn)
                                *MaxColumn = Column;
                        }
                        if(!gpuIsReceptive(NeighbourCell))
                        {
                            NextGrid.Cells[Neighbour.Row * NextGrid.Size + Neighbour.Column].Type = BOUNDARY;
                        }
                    }
                }

            }
        }
    }
}

__global__ void
IterationGPU(grid CurrentGrid, grid NextGrid, u32 *MaxColumn, f32 *Alpha, f32 *Beta, f32 *Gamma)
{
    u32 LocalID = threadIdx.x;
    u32 GlobalID = blockIdx.x * blockDim.x + threadIdx.x;
    u32 Row = GlobalID / CurrentGrid.Size;
    u32 Column = GlobalID % CurrentGrid.Size;

    if(GlobalID < CurrentGrid.Size*CurrentGrid.Size)
    {
        cell Cell = CurrentGrid.Cells[GlobalID];

        if(Cell.Type == EDGE)
        {
            CurrentGrid.Cells[GlobalID].Value = *Beta;
        }
        else
        {
            f32 NewWaterValue = 0.0f;
            if(gpuIsReceptive(Cell))
            {
                NewWaterValue += Cell.Value;

                f32 NeighbourDiffusion = 0.0f;
                for(u32 Direction = 0; Direction < 6; ++Direction)
                {
                    ivec2 Neighbour = gpuGridNeighbour(Row, Column, Direction);
                    if(Neighbour.Row < 0 || Neighbour.Row > CurrentGrid.Size || Neighbour.Column < 0 || Neighbour.Column > CurrentGrid.Size)
                    {
                        // NOTE(miha): Out of bounds, ignore it.
                    }
                    else
                    {
                        cell NeighbourCell = CurrentGrid.Cells[Neighbour.Row * CurrentGrid.Size + Neighbour.Column];
                        if(!gpuIsReceptive(NeighbourCell))
                        {
                            NeighbourDiffusion += NeighbourCell.Value;
                        }
                    }
                }
                NeighbourDiffusion /= 6.0f;
                NewWaterValue += (*Alpha/2.0f)*NeighbourDiffusion;
                NewWaterValue += *Gamma;
            }
            else
            {
                NewWaterValue += Cell.Value;
                f32 NeighbourDiffusion = 0.0f;
                for(u32 Direction = 0; Direction < 6; ++Direction)
                {
                    ivec2 Neighbour = gpuGridNeighbour(Row, Column, Direction);
                    if(Neighbour.Row < 0 || Neighbour.Row > CurrentGrid.Size || Neighbour.Column < 0 || Neighbour.Column > CurrentGrid.Size)
                    {
                        // NOTE(miha): Out of bounds, ignore it.
                    }
                    else
                    {
                        cell NeighbourCell = CurrentGrid.Cells[Neighbour.Row * CurrentGrid.Size + Neighbour.Column];
                        if(!gpuIsReceptive(NeighbourCell))
                        {
                            NeighbourDiffusion += NeighbourCell.Value;
                        }
                    }
                }
                NeighbourDiffusion /= 6.0f;
                NewWaterValue += (*Alpha/2.0f)*(NeighbourDiffusion - Cell.Value);
            }

            NextGrid.Cells[GlobalID].Value = NewWaterValue;
            if(NewWaterValue > 1.0f)
            {
                NextGrid.Cells[GlobalID].Type = FROZEN;
                for(u32 Direction = 0; Direction < 6; ++Direction)
                {
                    ivec2 Neighbour = gpuGridNeighbour(Row, Column, Direction);
                    if(Neighbour.Row < 0 || Neighbour.Row > CurrentGrid.Size || Neighbour.Column < 0 || Neighbour.Column > CurrentGrid.Size)
                    {
                        // NOTE(miha): Out of bounds, ignore it.
                    }
                    else
                    {
                        cell NeighbourCell = NextGrid.Cells[Neighbour.Row * NextGrid.Size + Neighbour.Column];
                        if(NeighbourCell.Type == EDGE)
                        {
                            if(Column > *MaxColumn)
                                *MaxColumn = Column;
                        }
                        if(!gpuIsReceptive(NeighbourCell))
                        {
                            NextGrid.Cells[Neighbour.Row * NextGrid.Size + Neighbour.Column].Type = BOUNDARY;
                        }
                    }
                }

            }
        }
    }
}

// NOTE(miha): We generate image with CPU and copy it to the GPU.

i32
main(i32 ArgumentCount, char *ArgumentValues[])
{
    f32 Alpha = atof(ArgumentValues[1]);
    f32 Beta = atof(ArgumentValues[2]);
    f32 Gamma = atof(ArgumentValues[3]);
    u32 Size = atoi(ArgumentValues[4]);
    i32 MaxIteration = atoi(ArgumentValues[5]);

    grid Grid = {};
    // CARE(miha): CellSize is the radius of a cell!
    Grid.CellSize = 10;
    // CARE(miha): Border is included into Grid.Size!
    Grid.Size = Size;
    // NOTE(miha): We have two grids; one for time t and one for time t+1.
    grid NewGrid = {};
    NewGrid.CellSize = Grid.CellSize;
    NewGrid.Size = Grid.Size;
    if(GenerateGrid(&Grid, Beta) && GenerateGrid(&NewGrid, Beta))
    {
        image Image = {};
        Image.ToPixelMultiplier = Grid.CellSize;
        Image.ChannelsPerPixel = 3;
        Image.Width = Grid.Size * 1.5f * Image.ToPixelMultiplier;
        Image.Height = Grid.Size * sqrtf(3) * Image.ToPixelMultiplier;
        u8 *ImagePixels = (u8 *)malloc(Image.Width*Image.Height*Image.ChannelsPerPixel);
        Image.Pixels = ImagePixels;

        b32 Running = 1;
        u32 Iteration = 0;
        u32 FromIterations = 0;

        // NOTE(miha): First iteration we do calculations in the 'NextGrid'.
        grid *CurrentGrid = &Grid;
        grid *NextGrid = &NewGrid;

        u32 BlockSize = 256;
        u32 GridSize = ((Grid.Size*Grid.Size)/BlockSize) + 1;

        cell *gpuCurrentGridCells;
        checkCudaErrors(hipMalloc(&gpuCurrentGridCells, Grid.Size*Grid.Size*sizeof(cell)));
        checkCudaErrors(hipMemcpy(gpuCurrentGridCells, CurrentGrid->Cells, Grid.Size*Grid.Size*sizeof(cell), hipMemcpyHostToDevice));
        CurrentGrid->Cells = gpuCurrentGridCells;

        cell *gpuNextGridCells;
        checkCudaErrors(hipMalloc(&gpuNextGridCells, NewGrid.Size*NewGrid.Size*sizeof(cell)));
        checkCudaErrors(hipMemcpy(gpuNextGridCells, NextGrid->Cells, NewGrid.Size*NewGrid.Size*sizeof(cell), hipMemcpyHostToDevice));
        NextGrid->Cells = gpuNextGridCells;

        u32 *gpuMaxColumn;
        checkCudaErrors(hipMalloc(&gpuMaxColumn, sizeof(u32)));
        checkCudaErrors(hipMemset(gpuMaxColumn, 0, sizeof(u32)));

        f32 *gpuAlpha;
        checkCudaErrors(hipMalloc(&gpuAlpha, sizeof(f32)));
        checkCudaErrors(hipMemcpy(gpuAlpha, &Alpha, sizeof(f32), hipMemcpyHostToDevice));

        f32 *gpuBeta;
        checkCudaErrors(hipMalloc(&gpuBeta, sizeof(f32)));
        checkCudaErrors(hipMemcpy(gpuBeta, &Beta, sizeof(f32), hipMemcpyHostToDevice));

        f32 *gpuGamma;
        checkCudaErrors(hipMalloc(&gpuGamma, sizeof(f32)));
        checkCudaErrors(hipMemcpy(gpuGamma, &Gamma, sizeof(f32), hipMemcpyHostToDevice));

        hipEvent_t Start, Stop;
        hipEventCreate(&Start);
        hipEventCreate(&Stop);
        hipEventRecord(Start);

        while(Running)
        {
            printf("iteration: %d\n", Iteration);
            u32 MaxColumn = 0;

            // TODO(miha): Call CUDA to calculate one iteration.
            // have to pass poiter to current grid on gpu, pointer to next grid on gpu, grid size
            IterationGPU<<<GridSize, BlockSize>>>(*CurrentGrid, *NextGrid, gpuMaxColumn, gpuAlpha, gpuBeta, gpuGamma);
            hipDeviceSynchronize();
            getLastCudaError("HistogramGPU() execution failed\n");
            checkCudaErrors(hipMemcpy(&MaxColumn, gpuMaxColumn, sizeof(u32), hipMemcpyDeviceToHost));

            // NOTE(miha): Switch grids.
            grid *Temp = NextGrid;
            NextGrid = CurrentGrid;
            CurrentGrid = Temp;

            if(MaxIteration == -1)
            {
                if(MaxColumn == CurrentGrid->Size-2)
                    Running = 0;
            }
            else
            {
                if(Iteration > (u32)MaxIteration)
                    Running = 0;
            }
            //if(FromIterations && Iteration - FromIterations > 20)
            //    Running = 0;
            //if(Iteration > 1000)
            //    Running = 0;
            //Running = 0;

#if defined(SAVE_DURING_ITERATIONS)
            if(Iteration % SAVE_DURING_ITERATIONS_INTERVAL == 0)
            {
                char FileNameBuffer[256] = {};
                snprintf(FileNameBuffer, 256, "out%d.png", Iteration / SAVE_DURING_ITERATIONS_INTERVAL);
                //PrintGrid(CurrentGrid);
                DrawGrid(CurrentGrid, &Image);
                stbi_write_png(FileNameBuffer, Image.Width, Image.Height, Image.ChannelsPerPixel, Image.Pixels, Image.Width*Image.ChannelsPerPixel);
            }
#endif
            Iteration++;
        }

        hipEventRecord(Stop);
        hipEventSynchronize(Stop);

        f32 Milliseconds = 0;
        hipEventElapsedTime(&Milliseconds, Start, Stop);
        printf("Time: %0.3f milliseconds \n", Milliseconds);

        cell *Cells = (cell *)malloc(Grid.Size*Grid.Size*sizeof(cell));
        memset(Cells, 0, Grid.Size*Grid.Size*sizeof(cell));
        checkCudaErrors(hipMemcpy(Cells, gpuCurrentGridCells, Grid.Size*Grid.Size*sizeof(cell), hipMemcpyDeviceToHost));
        CurrentGrid->Cells = Cells;
        DrawGrid(CurrentGrid, &Image);
        stbi_write_png("out_cuda.png", Image.Width, Image.Height, Image.ChannelsPerPixel, Image.Pixels, Image.Width*Image.ChannelsPerPixel);
    }
    else
    {
    }
}
